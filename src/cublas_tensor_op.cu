// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 20:42:28 on Sun, Feb 12, 2023
//
// Description: cublas tensor op back2back hgemm

#include "common.h"
#include "matrix.h"

hipblasHandle_t getCublasTensorOpHandle() {
    hipblasHandle_t handle = nullptr;
    HGEMM_CHECK_CUBLAS_ERROR(hipblasCreate(&handle));
    HGEMM_CHECK_CUBLAS_ERROR(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    return handle;
}

void cublasTensorOp(half *A, half *B, half *C, half *D, size_t M, size_t L, size_t N, size_t K) {
    static hipblasHandle_t handle = getCublasTensorOpHandle();
    static half alpha = 1.0;
    static half beta = 0.0;

    static Matrix *P = new Matrix(M, N, "Matrix P");

    HGEMM_CHECK_CUBLAS_ERROR(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A,
                                          HIP_R_16F, K, &beta, P->getDevPtr(), HIP_R_16F, N, HIPBLAS_COMPUTE_16F,
                                          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    HGEMM_CHECK_CUBLAS_ERROR(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, L, M, N, &alpha, C, HIP_R_16F, N,
                                          P->getDevPtr(), HIP_R_16F, N, &beta, D, HIP_R_16F, L, HIPBLAS_COMPUTE_16F,
                                          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}
